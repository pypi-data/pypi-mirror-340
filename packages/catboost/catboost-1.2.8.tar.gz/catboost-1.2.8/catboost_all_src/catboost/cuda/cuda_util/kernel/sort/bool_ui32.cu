#include <catboost/cuda/cuda_util/kernel/sort_templ.cuh>

namespace NKernel {
    template hipError_t RadixSort(bool* keys, ui32* values, ui32 size, TRadixSortContext& context, TCudaStream stream);
}
