#include <catboost/cuda/cuda_util/kernel/sort_templ.cuh>

namespace NKernel {
    template hipError_t RadixSort(ui64* keys, uchar* values, ui32 size, TRadixSortContext& context,  TCudaStream stream);
}
