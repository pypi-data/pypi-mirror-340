#include "hip/hip_runtime.h"
#include <catboost/cuda/cuda_util/kernel/sort_templ.cuh>

namespace NKernel {
    template hipError_t RadixSort(float * keys, uint2* values, ui32 size, TRadixSortContext& context,  TCudaStream stream);
}
