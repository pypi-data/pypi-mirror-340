#include <catboost/cuda/cuda_util/kernel/sort_templ.cuh>

namespace NKernel {
    template hipError_t RadixSort(float* keys, ui16* values, ui32 size, TRadixSortContext& context, TCudaStream stream);
}
