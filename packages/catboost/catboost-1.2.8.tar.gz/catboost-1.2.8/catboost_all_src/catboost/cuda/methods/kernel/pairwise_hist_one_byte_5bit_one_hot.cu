#include "hip/hip_runtime.h"
#include "pairwise_hist_one_byte_5bit.cuh"

using namespace cooperative_groups;

namespace NKernel {

    template
    void ComputePairwiseHistogramOneByte5BitsImpl<true>(const TCFeature* features, const TCFeature* featuresCpu,
                                                        const ui32 featureCount,
                                                        const ui32 fiveBitsFeatureCount,
                                                        const ui32* compressedIndex,
                                                        const uint2* pairs, ui32 pairCount,
                                                        const float* weight,
                                                        const TDataPartition* partition,
                                                        ui32 partCount,
                                                        ui32 histLineSize,
                                                        bool fullPass,
                                                        float* histogram,
                                                        int parallelStreams,
                                                        TCudaStream stream);


}
